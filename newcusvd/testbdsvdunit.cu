#include "hip/hip_runtime.h"
/************************************************************************************************
* Implementing Singular Value Decomposition on GPU using CUDA using algorithm                   *
* given in IPDPS '09 paper "Singular Value Decomposition on GPU using CUDA"                     *
*                                                                                               *
* Copyright (c) 2009 International Institute of Information Technology, Hyderabad.              *
* All rights reserved.                                                                          *
*                                                                                               *
* Permission to use, copy, modify and distribute this software and its documentation for        *
* educational purpose is hereby granted without fee, provided that the above copyright         *
* notice and this permission notice appear in all copies of this software and that you do       *
* not sell the software.                                                                        *
*                                                                                               *
* THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND, EXPRESS, IMPLIED OR        *
* OTHERWISE.                                                                                    *
*                                                                                               *
* Created by Sheetal Lahabar.                                                                   *
* **********************************************************************************************/

/************************************************************************************************
 Copyright (c) 1992-2007 The University of Tennessee.  All rights reserved.
 Contributors:
    * Sergey Bochkanov (ALGLIB project). Translation from FORTRAN to
      pseudocode.

 See subroutines comments for additional copyrights.
 Redistribution and use in source and binary forms, with or without
 modification, are permitted provided that the following conditions are
 met:

 - Redistributions of source code must retain the above copyright
 notice, this list of conditions and the following disclaimer.

 - Redistributions in binary form must reproduce the above copyright
 notice, this list of conditions and the following disclaimer listed
 in this license in the documentation and/or other materials
 provided with the distribution.

 - Neither the name of the copyright holders nor the names of its
 contributors may be used to endorse or promote products derived from
 this software without specific prior written permission.

 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
 A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

************************************************************************************************/

#ifndef _TESTBDSVDUNIT_CU_
#define _TESTBDSVDUNIT_CU_

#include <stdio.h>
#include <stdlib.h>
#include "hipblas.h"
#include <cutil.h>
#include <sys/timeb.h>
#include <time.h>

#include "testbdsvdunit.h"
#include "bdsvd.h"

struct timeb tt1;
void fillidentity(ap::real_2d_array& a, int n);
void fillsparsede(ap::real_1d_array& d,
     ap::real_1d_array& e,
     int n,
     double sparcity);
void getbdsvderror(const ap::real_1d_array& d,
     const ap::real_1d_array& e,
     int n,
     bool isupper,
     const ap::real_2d_array& u,
     const ap::real_2d_array& c,
     const ap::real_1d_array& w,
     const ap::real_2d_array& vt,
     double& materr,
     double& orterr,
     bool& wsorted);
void testbdsvdproblem(const ap::real_1d_array& d,
     const ap::real_1d_array& e,
     int n,
     double& materr,
     double& orterr,
     bool& wsorted,
     bool& wfailed);

/*************************************************************************
Testing bidiagonal SVD decomposition subroutine
*************************************************************************/

bool testbdsvd(int M, int N, bool silent, float *dU, float *dV, float *dC, float *dd, double *diagonal, double *superdiag)
{
    ap::real_1d_array d;
    ap::real_1d_array e;
    ap::real_2d_array mempty;
    
    int n;
    int maxn;
    int i;
    bool failcase;
    
    maxn = N;
    int maxm = M;

    d.setbounds(0, maxn-1);
    e.setbounds(0, maxn-2);

    n = N;
    int m = maxm;
    
    for(i=0; i < n; i++)
	    d(i) = diagonal[i];

    for(i=0; i < n-1; i++)
	    e(i) = superdiag[i];	

    int n1 = 0, n2 = 0, n3 = 0, n4 = 0;

    printf("Computing Diagonal matrix Sigma\n");
    ftime(&tt1);
    n1 = tt1.time;
    n2 = tt1.millitm;

    failcase = rmatrixbdsvd(d, e, n, true, true, m, n, dU, dV, dC, dd);
    hipDeviceSynchronize();

    ftime(&tt1);
    n3 = tt1.time;
    n4 = tt1.millitm;

    printf("Time required for diagonalization %d %d \n", (n3-n1), (n4-n2));
    for(i=0; i < n; i++)
	    diagonal[i] = d(i);

    return failcase;

    //
    // special case: zero divide matrix
    // unfixed LAPACK routine should fail on this problem
    //

    /*    
	  n = 7;
	  d(0) = -6.96462904751731892700e-01;
	  d(1) = 0.00000000000000000000e+00;
	  d(2) = -5.73827770385971991400e-01;
	  d(3) = -6.62562624399371191700e-01;
	  d(4) = 5.82737148001782223600e-01;
	  d(5) = 3.84825263580925003300e-01;
	  d(6) = 9.84087420830525472200e-01;
	  e(0) = -7.30307931760612871800e-02;
	  e(1) = -2.30079042939542843800e-01;
	  e(2) = -6.87824621739351216300e-01;
	  e(3) = -1.77306437707837570600e-02;
	  e(4) = 1.78285126526551632000e-15;
	  e(5) = -4.89434737751289969400e-02;
	  rmatrixbdsvd(d, e, n, true, false, mempty, 0, mempty, 0, mempty, 0);
    */

    //
    // zero matrix, several cases
    //

   /*
    for(i = 0; i <= maxn-1; i++)
    {
	    d(i) = 0;
    }
    for(i = 0; i <= maxn-2; i++)
    {
	    e(i) = 0;
    }
    for(n = 1; n <= maxn; n++)
    {
	    testbdsvdproblem(d, e, n, materr, orterr, wsorted, wfailed);
    }

    //
    // Dense matrix
    //
    for(n = 1; n <= maxn; n++)
    {
	    for(pass = 1; pass <= 10; pass++)
	    {
		    for(i = 0; i <= maxn-1; i++)
		    {
			    d(i) = 2*ap::randomreal()-1;
		    }
		    for(i = 0; i <= maxn-2; i++)
		    {
			    e(i) = 2*ap::randomreal()-1;
		    }
		    testbdsvdproblem(d, e, n, materr, orterr, wsorted, wfailed);
	    }
    }

    //
    // Sparse matrices, very sparse matrices, incredible sparse matrices
    //
    for(n = 1; n <= maxn; n++)
    {
	    for(pass = 1; pass <= 10; pass++)
	    {
		    fillsparsede(d, e, n, 0.5);
		    testbdsvdproblem(d, e, n, materr, orterr, wsorted, wfailed);
		    fillsparsede(d, e, n, 0.8);
		    testbdsvdproblem(d, e, n, materr, orterr, wsorted, wfailed);
		    fillsparsede(d, e, n, 0.9);
		    testbdsvdproblem(d, e, n, materr, orterr, wsorted, wfailed);
		    fillsparsede(d, e, n, 0.95);
		    testbdsvdproblem(d, e, n, materr, orterr, wsorted, wfailed);
	    }
    }

    //
    // report
    //

    failr = double(failcount)/double(succcount+failcount);
    waserrors = materr>threshold||orterr>threshold||!wsorted||failr>failthreshold;

    if( !silent )
    {
	    printf("TESTING BIDIAGONAL SVD DECOMPOSITION\n");
	    printf("SVD decomposition error:                 %5.3le\n",
			    double(materr));
	    printf("SVD orthogonality error:                 %5.3le\n",
			    double(orterr));
	    printf("Singular values order:                   ");
	    if( wsorted )
	    {
		    printf("OK\n");
	    }
	    else
	    {
		    printf("FAILED\n");
	    }
	    printf("Always converged:                        ");
	    if( !wfailed )
	    {
		    printf("YES\n");
	    }
	    else
	    {
		    printf("NO\n");
		    printf("Fail ratio:                              %5.3lf\n",
				    double(failr));
	    }
	    printf("Fail matrix test:                        ");
	    if( !failcase )
	    {
		    printf("AS EXPECTED\n");
	    }
	    else
	    {
		    printf("CONVERGED (UNEXPECTED)\n");
	    }
	    printf("Threshold:                               %5.3le\n",
			    double(threshold));
	    if( waserrors )
	    {
		    printf("TEST FAILED\n");
	    }
	    else
	    {
		    printf("TEST PASSED\n");
	    }
	    printf("\n\n");
    }
    result = !waserrors;
    return result;
    */
}

void fillidentity(ap::real_2d_array& a, int n)
{
/*	int i;
	int j;
	a.setbounds(0, n-1, 0, n-1);
	for(i = 0; i <= n-1; i++)
	{
		for(j = 0; j <= n-1; j++)
		{
			if( i==j )
			{
				a(i,j) = 1;
			}
			else
			{
				a(i,j) = 0;
			}
		}
	}
*/
}

void fillsparsede(ap::real_1d_array& d,
		ap::real_1d_array& e,
		int n,
		double sparcity)
{
/*	int i;
	int j;

	d.setbounds(0, n-1);
	e.setbounds(0, ap::maxint(0, n-2));
	for(i = 0; i <= n-1; i++)
	{
		if( ap::randomreal()>=sparcity )
		{
			d(i) = 2*ap::randomreal()-1;
		}
		else
		{
			d(i) = 0;
		}
	}
	for(i = 0; i <= n-2; i++)
	{
		if( ap::randomreal()>=sparcity )
		{
			e(i) = 2*ap::randomreal()-1;
		}
		else
		{
			e(i) = 0;
		}
	}
*/
}

void getbdsvderror(const ap::real_1d_array& d,
		const ap::real_1d_array& e,
		int n,
		bool isupper,
		const ap::real_2d_array& u,
		const ap::real_2d_array& c,
		const ap::real_1d_array& w,
		const ap::real_2d_array& vt,
		double& materr,
		double& orterr,
		bool& wsorted)
{
/*      int i;
        int j;
        int k;
        double locerr;
        double sm;

	//
	// decomposition error
	//

	locerr = 0;
	for(i = 0; i <= n-1; i++)
	{
	for(j = 0; j <= n-1; j++)
	{
	sm = 0;
	for(k = 0; k <= n-1; k++)
	{
	sm = sm+w(k)*u(i,k)*vt(k,j);
	}
	if( isupper )
	{
	if( i==j )
	{
	locerr = ap::maxreal(locerr, fabs(d(i)-sm));
	}
	else
	{
	if( i==j-1 )
	{
	locerr = ap::maxreal(locerr, fabs(e(i)-sm));
	}
	else
	{
	locerr = ap::maxreal(locerr, fabs(sm));
	}
	}
	}
	else
	{
	if( i==j )
	{
	locerr = ap::maxreal(locerr, fabs(d(i)-sm));
	}
	else
	{
	if( i-1==j )
	{
	locerr = ap::maxreal(locerr, fabs(e(j)-sm));
	}
	else
	{
	locerr = ap::maxreal(locerr, fabs(sm));
	}
	}
	}
	}
	}
	materr = ap::maxreal(materr, locerr);

	//
	// check for C = U'
	// we consider it as decomposition error
	//

	locerr = 0;
	for(i = 0; i <= n-1; i++)
	{
		for(j = 0; j <= n-1; j++)
		{
		locerr = ap::maxreal(locerr, fabs(u(i,j)-c(j,i)));
		}
	}	
	materr = ap::maxreal(materr, locerr);

	//
	// orthogonality error
	//

	locerr = 0;
	for(i = 0; i <= n-1; i++)
	{
		for(j = i; j <= n-1; j++)
		{
			sm = ap::vdotproduct(u.getcolumn(i, 0, n-1), u.getcolumn(j, 0, n-1));
			if( i!=j )
			{
				locerr = ap::maxreal(locerr, fabs(sm));
			}
			else
			{
				locerr = ap::maxreal(locerr, fabs(sm-1));
			}
			sm = ap::vdotproduct(&vt(i, 0), &vt(j, 0), ap::vlen(0,n-1));
			if( i!=j )
			{
				locerr = ap::maxreal(locerr, fabs(sm));
			}
			else
			{
				locerr = ap::maxreal(locerr, fabs(sm-1));
			}
		}
	}
	orterr = ap::maxreal(orterr, locerr);

	//
	// values order error
	//

	for(i = 1; i <= n-1; i++)
	{
		if( w(i)>w(i-1) )
		{
			wsorted = false;
		}
	}
*/
}


void testbdsvdproblem(const ap::real_1d_array& d,
		const ap::real_1d_array& e,
		int n,
		double& materr,
		double& orterr,
		bool& wsorted,
		bool& wfailed)
{
 /*   
		   ap::real_2d_array u;
		   ap::real_2d_array vt;
		   ap::real_2d_array c;
		   ap::real_1d_array w;
		   int i;
		   int j;
		   int k;
		   double v;
		   double mx;

		   mx = 0;
		   for(i = 0; i <= n-1; i++)
		   {
			   if( fabs(d(i))>mx )
		       	   {
 				   mx = fabs(d(i));
			   }
		   }
		   
		   for(i = 0; i <= n-2; i++)
		   {
		   	if( fabs(e(i))>mx )
		   	{
		   		mx = fabs(e(i));
		   	}
		   }
		   if( mx==0 )
		   {
		  	 mx = 1;
		   }

	//
	// Upper BDSVD tests
	//

	w.setbounds(0, n-1);
	fillidentity(u, n);
	fillidentity(vt, n);
	fillidentity(c, n);
	for(i = 0; i <= n-1; i++)
	{
		w(i) = d(i);
	}
	if( !rmatrixbdsvd(w, e, n, true, false, u, n, c, n, vt, n) )
	{
		failcount = failcount+1;
		wfailed = true;
		return;
	}
	getbdsvderror(d, e, n, true, u, c, w, vt, materr, orterr, wsorted);
	fillidentity(u, n);
	fillidentity(vt, n);
	fillidentity(c, n);
	for(i = 0; i <= n-1; i++)
	{
		w(i) = d(i);
	}
	if( !rmatrixbdsvd(w, e, n, true, true, u, n, c, n, vt, n) )
	{
		failcount = failcount+1;
		wfailed = true;
		return;
	}
	getbdsvderror(d, e, n, true, u, c, w, vt, materr, orterr, wsorted);

	//
	// Lower BDSVD tests
	//
	w.setbounds(0, n-1);
	fillidentity(u, n);
	fillidentity(vt, n);
	fillidentity(c, n);
	for(i = 0; i <= n-1; i++)
	{
		w(i) = d(i);
	}
	if( !rmatrixbdsvd(w, e, n, false, false, u, n, c, n, vt, n) )
	{
		failcount = failcount+1;
		wfailed = true;
		return;
	}
	getbdsvderror(d, e, n, false, u, c, w, vt, materr, orterr, wsorted);

	fillidentity(u, n);
	fillidentity(vt, n);
	fillidentity(c, n);

	for(i = 0; i <= n-1; i++)
	{
		w(i) = d(i);
	}
	if( !rmatrixbdsvd(w, e, n, false, true, u, n, c, n, vt, n) )
	{
		failcount = failcount+1;
		wfailed = true;
		return;
	}
	getbdsvderror(d, e, n, false, u, c, w, vt, materr, orterr, wsorted);

	//
	// update counter
	//
	succcount = succcount+1;
*/
}

/*************************************************************************
  Silent unit test
 *************************************************************************/
bool testbdsvdunit_test_silent()
{
	bool result = 1;
	//result = testbdsvd(true);
	return result;
}


/*************************************************************************
  Unit test
 *************************************************************************/
bool testbdsvdunit_test()
{
	bool result = 1;
	//result = testbdsvd(false);
	return result;
}
#endif

