#include "hip/hip_runtime.h"
/************************************************************************************************
* Implementing Singular Value Decomposition on GPU using CUDA using algorithm                   *
* given in IPDPS '09 paper "Singular Value Decomposition on GPU using CUDA"                     *
*                                                                                               *
* Copyright (c) 2009 International Institute of Information Technology, Hyderabad.              *
* All rights reserved.                                                                          *
*                                                                                               *
* Permission to use, copy, modify and distribute this software and its documentation for        *
* educational purpose is hereby granted without fee, provided that the above copyright          *
* notice and this permission notice appear in all copies of this software and that you do       *
* not sell the software.                                                                        *
*                                                                                               *
* THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND, EXPRESS, IMPLIED OR        *
* OTHERWISE.                                                                                    *
*                                                                                               *
* Created by Sheetal Lahabar.                                                                   *
************************************************************************************************/

/*************************************************************************
Copyright (c) 1992-2007 The University of Tennessee.  All rights reserved.

Contributors:
    * Sergey Bochkanov (ALGLIB project). Translation from FORTRAN to
      pseudocode.

See subroutines comments for additional copyrights.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are
met:

- Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

- Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer listed
in this license in the documentation and/or other materials
provided with the distribution.

- Neither the name of the copyright holders nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
"AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *************************************************************************/

#include "hipblas.h"
#include <cutil.h>
#include "rotations.h"

#ifndef _ROTATIONS_CU_
#define _ROTATIONS_CU_

#define xBS 64
#define BS 64
#define CS 192

/*************************************************************************
 * Kernel applies a series of elementary rotations to a matrix in the 
 * forward(top to bottom) direction.  
 * 
 * The algorithm multiplies the matrix by a sequence of rotation
 * transformations which are given by arrays d_c and d_s. The vectors d_c 
 * and d_s of length (M2-M1+1 or N2-N1+1) fit in the shared memory. 
 * 
 * Not the whole matrix but only a part of it is transformed (rows from M1
 * to M2 of dU or rows from N1 to N2 of dV). Only the elements of this submatrix are 
 * changed.
 * 
 * Input parameters:
 *     g_data      - Matrix to be transformed (dU or dV)
 *     d_c, d_s    - Coefficient vectors on the device
 * 
 * Output parameters:
 *     g_data      - Transformed matrix
 * 
 * Utility subroutine.
 *************************************************************************/

__global__ void forwardonce(float *g_data, float *d_c, float *d_s, int width, int height)
{
	__shared__ float a[BS];
	__shared__ float b[BS];

	__shared__ float c[CS];
	__shared__ float s[CS];

	int i=0;

	int tx   = threadIdx.x;
	int bx   = blockIdx.x;
	int Dimx = blockDim.x;

	float  ctemp=1, stemp=1;

	int    datacopy = CS / Dimx;

	int    ind      = __mul24(bx , Dimx) + tx;

	for(i=0; i < datacopy; i++)
	{
		c[ i * BS + tx ] = d_c[ i * BS + tx ];
		s[ i * BS + tx ] = d_s[ i * BS + tx ];
		__syncthreads();
	}

	a[tx]    = g_data[ind];
	__syncthreads();
	float a1, a2;
	float a1temp, a2temp;

	for(i=0 ; i < height-1; i++)
	{
		ctemp = c[i];
		stemp = s[i];

		b[tx] = g_data[ind + (i+1) * width];
		__syncthreads();

		a1temp = a[tx];
		a2temp = b[tx];

		a1     = ctemp * a1temp + stemp * a2temp;
		a2     = ctemp * a2temp - stemp * a1temp;
		__syncthreads();

		g_data[ind + (i) * width] = a1;
		__syncthreads();

		a[tx] = a2;
		__syncthreads();
	}

	g_data[ind + (i) * width] = a2;
	__syncthreads();
}

/****************************************************************************
 * Kernel applies a series of elementary rotations to a matrix in the 
 * backward(bottom to top) direction.  
 *
 * The algorithm multiplies the matrix by a sequence of rotation
 * transformations which are given by arrays d_c and d_s. The vectors d_c 
 * and d_s of length (M2-M1+1 or N2-N1+1) fit in the shared memory. 
 * 
 * Not the whole matrix but only a part of it is transformed (rows from M1
 * to M2 of dU and rows from N1 to N2 of dV). Only the elements of 
 * this submatrix are changed.
 * 
 * Input parameters:
 *    g_data      - Matrix to be transformed dU or dV
 *    d_c, d_s    - Coefficient vectors
 * 
 * Output parameters:
 *    g_data      - Transformed matrix
 *  
 * Utility subroutine.
 ****************************************************************************/

__global__ void backwardonce(float *g_data, float* d_c, float *d_s, int width, int height)
{
	__shared__ float a[BS];
	__shared__ float b[BS];

	__shared__ float c[CS];
	__shared__ float s[CS];

	int i=0;

	int tx   = threadIdx.x;
	int bx   = blockIdx.x;
	int Dimx = blockDim.x;

	float  ctemp=1, stemp=1;

	int    datacopy = CS / Dimx;

	int    ind      = __mul24(bx , Dimx) + tx;

	for(i=0; i < datacopy; i++)
	{
		c[i*BS + tx] = d_c[height - 2 - i*BS - tx];
		s[i*BS + tx] = d_s[height - 2 - i*BS - tx];
		__syncthreads();
	}

	a[tx]    = g_data[ind];
	__syncthreads();

	float a1, a2;
	float a1temp, a2temp;

	for(i=0 ; i < height-1; i++)
	{
		ctemp = c[i];
		stemp = s[i];

		b[tx] = g_data[ind - (i+1) * width];
		__syncthreads();

		a1temp = b[tx];
		a2temp = a[tx];

		a1     = ctemp * a1temp + stemp * a2temp;
		a2     = ctemp * a2temp - stemp * a1temp;
		__syncthreads();

		g_data[ind - i * width] = a2;
		__syncthreads();

		a[tx] = a1;
		__syncthreads();
	}

	g_data[ind - i * width] = a1;
	__syncthreads();

}

/*************************************************************************
 * Kernel applies a series of elementary rotations to a matrix in the 
 * backward(bottom to top) direction.  
 * 
 * The algorithm multiplies the matrix by a sequence of rotation
 * transformations which are given by arrays d_c and d_s. The vectors d_c 
 * and d_s of length (M2-M1+1 or N2-N1+1) do not fit in the shared memory.
 * They are loaded in batches. 
 * 
 * Not the whole matrix but only a part of it is transformed (rows from M1
 * to M2, rows from N1 to N2). Only the elements of this submatrix are 
 * changed.
 * 
 * Input parameters:
 * g_data      - Matrix to be transformed
 * d_c, d_s    - Coefficient vectors
 * 
 * Output parameters:
 * g_data is transformed.
 *  
 * Utility subroutine.
 ****************************************************************************/

__global__ void backward(float *g_data, float *d_c, float *d_s, int width, int height)
{
	__shared__ float a[BS];
	__shared__ float b[BS];

	__shared__ float c[CS];
	__shared__ float s[CS];

	int i=0, j=0, k=0;

	int tx = threadIdx.x;
	int bx = blockIdx.x;
	int Dimx = blockDim.x;

	int elements = (512 - (Dimx * 2))/2;
	float ctemp, stemp;
	
	int ind = __mul24(bx, Dimx) + tx;
	float a1, a2, flag = 0;
	int calls;
	float a1temp, a2temp;

        int datacopy = CS / Dimx;

        if((height - 1) % elements == 0)
        {
                flag = 0;
        }
        else
        {
                flag = 1;
        }

        calls = (height - 1) / elements;

        a[tx] = g_data[ind];
        __syncthreads();

	for(i=0 ; i < calls ; i++)
        {
                for(k=0 ; k < datacopy ; k++)
                {
                        c[k * BS + tx] = d_c[ height - 2 - i * CS - k * BS - tx];
                        __syncthreads();
                        s[k * BS + tx] = d_s[ height - 2 - i * CS - k * BS - tx];
                        __syncthreads();
                }
                for(j=0; j < elements ; j++)
                {
                        ctemp = c[j];
                        stemp = s[j];

                        b[tx] = g_data[ind - i * CS * width - (j+1) * width];
                        __syncthreads();

                        a1temp = b[tx];
                        a2temp = a[tx];
                        __syncthreads();

                        a1 = ctemp * a1temp + stemp * a2temp;
                        a2 = ctemp * a2temp - stemp * a1temp;
                        __syncthreads();

                        g_data[ind - i * CS * width - j * width] = a2;
                        __syncthreads();

                        a[tx] = a1;
                        __syncthreads();
                }
        }
	if(flag == 0)
        {
                g_data[ind - (i-1) * CS * width - j * width] = a1;
                __syncthreads();
        }
        else
        {
                for(k=0; k < datacopy; k++)
                {
                        c[ k * BS + tx ] = d_c[ height - 2 - i * CS - k * BS - tx ];
                        __syncthreads();
                        s[ k * BS + tx ] = d_s[ height - 2 - i * CS - k * BS - tx ];
                        __syncthreads();
                }
                for( j = 0; j < height - 1 - (calls * elements); j++)
                {
                        ctemp = c[j];
                        stemp = s[j];

                        b[tx] = g_data[ind - i * CS * width - (j + 1) * width];
                        __syncthreads();

                        a1temp = b[tx];
                        a2temp = a[tx];
			__syncthreads();

                        a1     = ctemp * a1temp + stemp * a2temp;
                        a2     = ctemp * a2temp - stemp * a1temp;
                        __syncthreads();

                        g_data[ ind - i * CS * width - j * width ] = a2;
                        __syncthreads();

                        a[tx] = a1;
                        __syncthreads();
                }

                g_data[ind - i * CS * width - j * width] = a1;
                __syncthreads();
        }
}

/*************************************************************************
 * Kernel applies a series of elementary rotations to a matrix in the 
 * forward(top to bottom) direction.  
 * 
 * The algorithm multiplies the matrix by a sequence of rotation
 * transformations which are given by arrays d_c and d_s. The vectors d_c 
 * and d_s of length (M2-M1+1 or N2-N1+1) do not fit in the shared memory. 
 * They are loaded in batches to the shared memory.
 * 
 * Not the whole matrix but only a part of it is transformed (rows from M1
 * to M2, rows from N1 to N2). Only the elements of this submatrix are 
 * changed.
 * 
 * Input parameters:
 *   g_data      - Matrix to be transformed
 *   d_c, d_s    - Coefficient vectors
 *  
 * Output parameters:
 *   g_data is transformed.
 *   
 *   Utility subroutine.
 **************************************************************************/


__global__ void forward(float *g_data, float *d_c, float *d_s, int width, int height)
{
	__shared__ float a[BS];
	__shared__ float b[BS];

	__shared__ float c[CS];
	__shared__ float s[CS];

	int i=0, j=0, k=0;

	int tx    =  threadIdx.x;
	int bx    =  blockIdx.x;
	int Dimx  =  blockDim.x;

	int elements = (512 - (Dimx * 2))/2;

	float ctemp, stemp;

	int ind = __mul24(bx, Dimx) + tx;

	float a1, a2, flag=0;
	int calls = 0;
	float a1temp, a2temp;

	int datacopy = CS / Dimx;

	if((height - 1) % elements == 0)
	{
		flag = 0;
	}
	else
	{
		flag = 1;
	}

	calls = (height - 1) / elements;

	a[tx] = g_data[ind];
	__syncthreads();

	for(i=0 ; i < calls ; i++)
	{
		for(k=0 ; k < datacopy ; k++)
		{
			c[k * BS + tx] = d_c[i * CS + k * BS + tx];
			__syncthreads();
			s[k * BS + tx] = d_s[i * CS + k * BS + tx];
			__syncthreads();
		}

		for(j=0; j < elements ; j++)
		{
			ctemp = c[j];
			stemp = s[j];

			b[tx] = g_data[ind + i * CS * width + (j+1) * width];
			__syncthreads();
			a1temp = a[tx];
			a2temp = b[tx];
			__syncthreads();

			a1 = ctemp * a1temp + stemp * a2temp;
			a2 = ctemp * a2temp - stemp * a1temp;
			__syncthreads();

			g_data[ind + i * CS * width + j * width] = a1;
			__syncthreads();

			a[tx] = a2;
			__syncthreads();
		}
	}
	if(flag == 0)
	{
		g_data[ind + (i-1) * CS * width + j * width] = a2;
		__syncthreads();
	}
	else
	{
		for(k=0; k < datacopy; k++)
		{
			c[ k * BS + tx ] = d_c[ i * CS + k * BS + tx ];
			__syncthreads();
			s[ k * BS + tx ] = d_s[ i * CS + k * BS + tx ];
			__syncthreads();
		}
		for( j = 0; j < height - 1 - (calls * elements); j++)
		{
			ctemp = c[j];
			stemp = s[j];

			b[tx] = g_data[ind + i * CS * width + (j + 1) * width];
			__syncthreads();

			a1temp = a[tx];
			a2temp = b[tx];
			a1     = ctemp * a1temp + stemp * a2temp;
			a2     = ctemp * a2temp - stemp * a1temp;
			__syncthreads();

			g_data[ ind + i * CS * width + j * width ] = a1;
			__syncthreads();

			a[tx] = a2;
			__syncthreads();
		}

		g_data[ind + i * CS * width + j * width] = a2;
		__syncthreads();
	}
}

/***********************************************************************************
 * Application of a sequence of  elementary rotations to a matrix
 *
 * The algorithm post-multiplies the matrix by a sequence of rotation
 * transformations which is given by arrays C and S. Depending on the value
 * of the IsForward parameter either 1 and 2, 3 and 4 and so on (if IsForward=true)
 * rows are rotated, or the rows N and N-1, N-2 and N-3 and so on are rotated.
 *
 * Not the whole matrix but only a part of it is transformed (rows from M1
 * to M2, columns from N1 to N2). Only the elements of this submatrix are changed.
 *
 * Input parameters:
 *     IsForward   -   the sequence of the rotation application.
 *         M1,M2   -   the range of rows to be transformed.
 *     N1, N2      -   the range of columns to be transformed.
 *     C,S         -   transformation coefficients.
 *                     Array whose index ranges within [1..N2-N1].
 *     WORK        -   working array whose index ranges within [M1..M2].
 *     mat         -   matrix to be transformed on the device    
 *     dC          -   temporary memory array on the device
 *     dd          -   temporaty memory array on the device
 *
 * Output parameters:
 *     mat         -   transformed matrix on the device
 *                                                                 
 * Utility subroutine.
 ************************************************************************************/

float* applyrotationsfromtheleft(bool isforward,
		int m1,
		int m2,
		int n1,
		int n2,
		const ap::real_1d_array& c,
		const ap::real_1d_array& s,
		ap::real_1d_array& work,
		float* mat, float* dC, float* dd)
{
	double ctemp;
	double stemp;

	if( m1>m2||n1>n2 )
	{
		return mat;
	}

	if( isforward )
	{
		if( n1!=n2 )
		{
			int i=0;
			int actlength = m2-m1+1;
			float *cs, *ss;

			dim3 threads;
			dim3 grid;

			cs = (float*)malloc(sizeof(float) * (actlength+xBS));
			ss = (float*)malloc(sizeof(float) * (actlength+xBS));

			threads.x = xBS;
			threads.y = 1;
		
			grid.x = (n2-n1+1)/xBS;
			grid.y = 1;

			for(i=1;i<m2-m1+1;i++)
                        {
                                ctemp = c(i);
                                stemp = s(i);

                                if(!(ctemp!=1 || stemp!=0))
                                {
                                        cs[i-1] = 1;
                                        ss[i-1] = 0;
                                }
                                else 
                                {
                                        cs[i-1]=ctemp;
                                        ss[i-1]=stemp;  
                                }
                        }       

			CUDA_SAFE_CALL(hipMemcpy(dd, &ss[0], (m2-m1+xBS)*sizeof(float), hipMemcpyHostToDevice));
			CUDA_SAFE_CALL(hipMemcpy(dC, &cs[0], (m2-m1+xBS)*sizeof(float), hipMemcpyHostToDevice));  

			if(m2-m1 <= CS)
			{
				forwardonce<<<grid, threads>>>(&mat[m1*(n2-n1+1)], dC, dd, n2-n1+1, m2-m1+1);
			}
			else
			{ 
				forward<<<grid, threads>>>(&mat[m1*(n2-n1+1)], dC, dd, n2-n1+1, m2-m1+1);
			}

			/*	for(j = m1; j <= m2-1; j++)
				{
					ctemp = c(j-m1+1);
					stemp = s(j-m1+1);
					if( ctemp!=1 || stemp!=0 )
					{
						jp1 = j+1;
						ap::vmove(&work(n1), &a(jp1, n1), ap::vlen(n1,n2), ctemp);
						ap::vsub(&work(n1), &a(j, n1), ap::vlen(n1,n2), stemp);
						ap::vmul(&a(j, n1), ap::vlen(n1,n2), ctemp);
						ap::vadd(&a(j, n1), &a(jp1, n1), ap::vlen(n1,n2), stemp);
						ap::vmove(&a(jp1, n1), &work(n1), ap::vlen(n1,n2));
					}
				}
			*/
			return mat;
		}
		else
		{
			//Special Case
			printf("Special case :(\n");

			/*
  			  for(j = m1; j <= m2-1; j++)
			  {
				  ctemp = c(j-m1+1);
				  stemp = s(j-m1+1);
				  if( ctemp!=1||stemp!=0 )
				  {
					  temp = a(j+1,n1);
					  a(j+1,n1) = ctemp*temp-stemp*a(j,n1);
					  a(j,n1) = stemp*temp+ctemp*a(j,n1);
			  	}
			  }
			*/
		}
		return mat;
	}
	else
	{
		if( n1!=n2 )
		{
			int i=0;
			int actlength = m2-m1+1;

			float *cs, *ss;
			cs = (float*)malloc(sizeof(float) * (actlength+xBS));
			ss = (float*)malloc(sizeof(float) * (actlength+xBS));

			dim3 threads;
			dim3 grid;

			grid.x = (n2-n1+1)/xBS;	
			grid.y = 1;

			threads.x = xBS;
			threads.y = 1;

			for(i=1;i<m2-m1+1;i++)
			{
				ctemp = c(i);
				stemp = s(i);

				if(!(ctemp!=1 || stemp!=0))
				{
					cs[i-1] = 1;
					ss[i-1] = 0;
				}
				else
				{
					cs[i-1]=ctemp;
					ss[i-1]=stemp;
				}

			}

			CUDA_SAFE_CALL(hipMemcpy(dd, &ss[0], (m2-m1+xBS)*sizeof(float), hipMemcpyHostToDevice));
			CUDA_SAFE_CALL(hipMemcpy(dC, &cs[0], (m2-m1+xBS)*sizeof(float), hipMemcpyHostToDevice));

			if(m2-m1 <= CS)	
			{
				backwardonce<<<grid, threads>>>(&mat[ m2 * (n2-n1+1) ], dC, dd, n2-n1+1, m2-m1+1);
			}
			else
			{
				backward<<<grid, threads>>>(&mat[ m2 * (n2-n1+1) ], dC, dd, n2-n1+1, m2-m1+1);
			}

			/*for(j = m2-1; j >= m1; j--)
			  {
			  ctemp = c(j-m1+1);
			  stemp = s(j-m1+1);

			  if( ctemp!=1||stemp!=0 )
			  {
			  jp1 = j+1;
			  ap::vmove(&work(n1), &a(jp1, n1), ap::vlen(n1,n2), ctemp);
			  ap::vsub(&work(n1), &a(j, n1), ap::vlen(n1,n2), stemp);
			  ap::vmul(&a(j, n1), ap::vlen(n1,n2), ctemp);
			  ap::vadd(&a(j, n1), &a(jp1, n1), ap::vlen(n1,n2), stemp);
			  ap::vmove(&a(jp1, n1), &work(n1), ap::vlen(n1,n2));
			  }

			  }*/

			return mat;
		}
		else
		{
			printf("Special case :(\n");
			//Special Case
			/*
			   for(j = m2-1; j >= m1; j--)
			   {
			   	ctemp = c(j-m1+1);
	  		   	stemp = s(j-m1+1);
	    		        if( ctemp!=1||stemp!=0 )
			        {
				   temp = a(j+1,n1);
				   a(j+1,n1) = ctemp*temp-stemp*a(j,n1);
				   a(j,n1) = stemp*temp+ctemp*a(j,n1);
			        }
			   }
			 */

		}

	}
	return mat;
}


/*************************************************************************
Application of a sequence of  elementary rotations to a matrix

The algorithm post-multiplies the matrix by a sequence of rotation
transformations which is given by arrays C and S. Depending on the value
of the IsForward parameter either 1 and 2, 3 and 4 and so on (if IsForward=true)
rows are rotated, or the rows N and N-1, N-2 and N-3 and so on are rotated.

Not the whole matrix but only a part of it is transformed (rows from M1
to M2, columns from N1 to N2). Only the elements of this submatrix are changed.

Input parameters:
    IsForward   -   the sequence of the rotation application.
    M1,M2       -   the range of rows to be transformed.
    N1, N2      -   the range of columns to be transformed.
    C,S         -   transformation coefficients.
                    Array whose index ranges within [1..N2-N1].
    WORK        -   working array whose index ranges within [M1..M2].
    mat         -   matrix to be transformed on the device
    dC          -   temporary memory array on the device
    dd          -   temporary memory array on the device
Output parameters:
    mat         -   transformed matrix on the device.

Utility subroutine.
*************************************************************************/

float* applyrotationsfromtheright(bool isforward,
     int m1,
     int m2,
     int n1,
     int n2,
     const ap::real_1d_array& c,
     const ap::real_1d_array& s,
     ap::real_1d_array& work, float* mat, float* dC, float* dd)
{
    return mat;

/*  int j;
    int jp1;
    double ctemp;
    double stemp;
    double temp;

    if( isforward )
    {
        if( m1!=m2 )
        {
            for(j = n1; j <= n2-1; j++)
            {
                ctemp = c(j-n1+1);
                stemp = s(j-n1+1);
                if( ctemp!=1||stemp!=0 )
                {
                    jp1 = j+1;
                    ap::vmove(work.getvector(m1, m2), a.getcolumn(jp1, m1, m2), ctemp);
                    ap::vsub(work.getvector(m1, m2), a.getcolumn(j, m1, m2), stemp);
                    ap::vmul(a.getcolumn(j, m1, m2), ctemp);
                    ap::vadd(a.getcolumn(j, m1, m2), a.getcolumn(jp1, m1, m2), stemp);
                    ap::vmove(a.getcolumn(jp1, m1, m2), work.getvector(m1, m2));
                }
            }
        }
        else
        {
	    //Special case
            for(j = n1; j <= n2-1; j++)
            {
                ctemp = c(j-n1+1);
                stemp = s(j-n1+1);
                if( ctemp!=1||stemp!=0 )
                {
                    temp = a(m1,j+1);
                    a(m1,j+1) = ctemp*temp-stemp*a(m1,j);
                    a(m1,j) = stemp*temp+ctemp*a(m1,j);
                }
            }
        }
    }
    else
    {
        if( m1!=m2 )
        {
            for(j = n2-1; j >= n1; j--)
            {
                ctemp = c(j-n1+1);
                stemp = s(j-n1+1);
                if( ctemp!=1||stemp!=0 )
                {
                    jp1 = j+1;
                    ap::vmove(work.getvector(m1, m2), a.getcolumn(jp1, m1, m2), ctemp);
                    ap::vsub(work.getvector(m1, m2), a.getcolumn(j, m1, m2), stemp);
                    ap::vmul(a.getcolumn(j, m1, m2), ctemp);
                    ap::vadd(a.getcolumn(j, m1, m2), a.getcolumn(jp1, m1, m2), stemp);
                    ap::vmove(a.getcolumn(jp1, m1, m2), work.getvector(m1, m2));
                }
            }
        }
        else
        {
	    //Special Case
            for(j = n2-1; j >= n1; j--)
            {
                ctemp = c(j-n1+1);
                stemp = s(j-n1+1);
                if( ctemp!=1||stemp!=0 )
                {
                    temp = a(m1,j+1);
                    a(m1,j+1) = ctemp*temp-stemp*a(m1,j);
                    a(m1,j) = stemp*temp+ctemp*a(m1,j);
                }
            }
        }
    }
    */
}


/*************************************************************************
The subroutine generates the elementary rotation, so that:

[  CS  SN  ]  .  [ F ]  =  [ R ]
[ -SN  CS  ]     [ G ]     [ 0 ]

CS**2 + SN**2 = 1
*************************************************************************/
void generaterotation(double f, double g, double& cs, double& sn, double& r)
{
    double f1;
    double g1;

    if( g==0 )
    {
        cs = 1;
        sn = 0;
        r = f;
    }
    else
    {
        if( f==0 )
        {
            cs = 0;
            sn = 1;
            r = g;
        }
        else
        {
            f1 = f;
            g1 = g;
            r = sqrt(ap::sqr(f1)+ap::sqr(g1));
            cs = f1/r;
            sn = g1/r;
            if( fabs(f)>fabs(g)&&cs<0 )
            {
                cs = -cs;
                sn = -sn;
                r = -r;
            }
        }
    }
}
#endif


