/************************************************************************************************
* Implementing Singular Value Decomposition on GPU using CUDA using algorithm 			*
* given in IPDPS '09 paper "Singular Value Decomposition on GPU using CUDA"			*
*												*
* Copyright (c) 2009 International Institute of Information Technology, Hyderabad.		*
* All rights reserved.										*
*												*
* Permission to use, copy, modify and distribute this software and its documentation for 	*
* educational purpose is hereby granted without fee, provided that the above copyright		*
* notice and this permission notice appear in all copies of this software and that you do 	*
* not sell the software.									*
*												*
* THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND, EXPRESS, IMPLIED OR	*
* OTHERWISE.											*
* 												*
* Created by Sheetal Lahabar.									*
************************************************************************************************/

#ifndef _CUDASVD_CU_
#define _CUDASVD_CU_

#include "cusvd.h"
#include "cubidiagonal.cu"
#include "cudiagonal.cu"

bool cusvd(int M, int N, float *d_A, float *d_MU, float *d_MVT, double *Sigma)
{
	int i = 0;
	bool result1, result2;

	float *dU, *dV;
	float *tempinitM, *tempinitN;
	double *diagonal, *superdiag;

	printf("Allocating resources, initializing variables\n");	

 	 diagonal =     (double*)malloc(sizeof(double)*N);
	superdiag = (double*)malloc(sizeof(double)*(N-1));

	tempinitM = (float*)malloc(sizeof(float)*M*M);
	tempinitN = (float*)malloc(sizeof(float)*N*N);

	for(i=0; i<M*M; i++)
		tempinitM[i]=0;

	for(i=0; i<N*N; i++)
		tempinitN[i]=0;

	for(i=0; i<M; i++)
		tempinitM[i*M+i]=1;

	for(i=0; i<N; i++)
		tempinitN[i*N+i]=1;

	CUDA_SAFE_CALL(hipMalloc((void**)&d_Q, M*M*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_P, N*N*sizeof(float)));
     
        CUDA_SAFE_CALL(hipMalloc((void**)&dU, M*M*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dV, N*N*sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpy(d_Q, tempinitM, M*M*sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_P, tempinitN, N*N*sizeof(float), hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(hipMemcpy(dU, tempinitM, M*M*sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dV, tempinitN, N*N*sizeof(float), hipMemcpyHostToDevice));

 	result1 = cubidiagonal(M, N, d_A, d_Q, d_P, diagonal, superdiag);
  	result2 =   cudiagonal(M, N, diagonal, superdiag, dU, dV);

	hipblasSgemm('n', 'n', M, M, M, 1, d_Q, M, dU, M, 0, d_MU, M);
        hipblasSgemm('t', 'n', N, N, N, 1, dV, N, d_P, N, 0, d_MVT, N);

/*	
        float *fsigma = (float*)malloc(sizeof(float)*N);
	for(int i=0; i<N; i++)	
	{
		Sigma[i] = diagonal[i];
		fsigma[i] = (float)Sigma[i];
	}

	float *d_middle, *zero, *check2;	
	for(int i=0; i<M*M; i++)
		zero[i] = 0;

	zero = (float*)malloc(sizeof(float)*M*M);
	check2 = (float*)malloc(sizeof(float)*M*M);

        CUDA_SAFE_CALL(hipMalloc((void**)&d_middle, M * M * sizeof(float)));
        CUDA_SAFE_CALL(hipMemcpy(d_middle, zero, M * M * sizeof(float), hipMemcpyHostToDevice));

        for(int i=0; i < M ;i++)
                CUDA_SAFE_CALL(hipMemcpy(&d_middle[i*M+i], &fsigma[i], sizeof(float), hipMemcpyHostToDevice));

        hipblasSgemm('n', 'n', N, N, N, 1,     d_MU, N, d_middle, N, 0, d_middle, N);
        hipblasSgemm('n', 'n', N, N, N, 1, d_middle, N,    d_MVT, N, 0, d_middle, N);

        CUDA_SAFE_CALL(hipMemcpy(check2, d_middle, sizeof(float)*N*N, hipMemcpyDeviceToHost));

        for(int i=0; i < M; i++)
                for(int j=0; j < N; j++)
                        printf("%f\n", check2[i*N+j]);
*/
	return result1 * result2;
}
#endif
