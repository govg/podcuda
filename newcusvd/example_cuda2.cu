/************************************************************************************************
* Implementing Singular Value Decomposition on GPU using CUDA using algorithm 			*
* given in IPDPS '09 paper "Singular Value Decomposition on GPU using CUDA"			*
*												*
* Copyright (c) 2009 International Institute of Information Technology, Hyderabad.		*
* All rights reserved.										*
*												*
* Permission to use, copy, modify and distribute this software and its documentation for 	*
* educational purpose is hereby granted without fee, provided that the above copyright		*
* notice and this permission notice appear in all copies of this software and that you do 	*
* not sell the software.									*
*												*
* THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND, EXPRESS, IMPLIED OR	*
* OTHERWISE.											*
* 												*
* Created by Sheetal Lahabar.									*
* Tested on CUDA 2.0                                                                            *
************************************************************************************************/

#ifndef _EXAMPLE_CU_
#define _EXAMPLE_CU_

#include "example.h"

//Include the below file in your main program
#include "cusvd.cu"

float *initialize(int ind)
{
	int i = 0, j = 0, l = 0;
        float *temp = (float*)malloc(sizeof(float) * ind * ind);

        for(i=0 ; i < ind ; i++)
        {
                for(j=0 ; j < ind ; j++)
                {
                        if(i==j)
                                temp[l++] = 1;
                        else
                                temp[l++] = 0;
                }
        }
        return temp;
}

int main(int argc, char** argv)
{
	bool   result;
	double *Sigma; 

	//M>=N and M and N are a multiple of 32
	int M = 512, N = 512;
	float *A, *U, *VT, *d_A, *d_U, *d_VT;

	//Step 1 - Read A in column major order 
	A = (float*)malloc(sizeof(float) * M * N);

	FILE *fp = fopen("data", "r");
	for(i=0 ; i < M * N ; i++)
	{
		fscanf(fp,"%f", &A[i]);
	}
	fclose(fp);

	//Step 2
	Sigma = (double*)malloc(sizeof(double)*N);

	//Step 3
	CUT_DEVICE_INIT(argc, argv);
	status = cublasInit();

	if(status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "Error in initialization");
		return EXIT_FAILURE;		
	}

	//Step 4
	
	status = CUDA_SAFE_CALL(cublasAlloc(M*N*sizeof(float), sizeof(float), (void**)&d_A));
	status = CUDA_SAFE_CALL(cublasAlloc(M*M*sizeof(float), sizeof(float), (void**)&d_U));	
	status = CUDA_SAFE_CALL(cublasAlloc(N*N*sizeof(float), sizeof(float), (void**)&d_VT));
	

	//Step 5
	U = initialize(M);
	VT = initialize(N);

	
	status = CUDA_SAFE_CALL(hipblasSetMatrix(M, N, sizeof(float),  A, M,  d_A, M));
	status = CUDA_SAFE_CALL(hipblasSetMatrix(M, N, sizeof(float),  U, M,  d_U, M));
	status = CUDA_SAFE_CALL(hipblasSetMatrix(M, N, sizeof(float), VT, M, d_VT, M));
	

	//Step 6
	timer = 0;
	CUT_SAFE_CALL(cutCreateTimer(&timer));
	CUT_SAFE_CALL(cutStartTimer(timer));
	
	result = cusvd(M, N, d_A, d_U, d_VT, Sigma);

	CUT_SAFE_CALL(cutStopTimer(timer));
	printf("SVD processing time: %f (ms)\n", cutGetTimerValue(timer));	
	CUT_SAFE_CALL(cutDeleteTimer(timer));

/*	
 	printf("Copy and print VT matrix\n");	
	CUDA_SAFE_CALL(hipMemcpy(VT, d_VT, sizeof(float)*N*N, hipMemcpyDeviceToHost));
	for(int i=0; i < N; i++)
		for(int j=0; j < N; j++)
			printf("%f\n", check2[i*N+j]);
*/

	//Step 7
	free(A);
	CUDA_SAFE_CALL(hipFree(d_A));
	CUDA_SAFE_CALL(hipFree(d_U));
	CUDA_SAFE_CALL(hipFree(d_VT));
	CUT_EXIT(argc, argv);
	return 0;
}
#endif
