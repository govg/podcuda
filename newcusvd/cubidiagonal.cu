#include "hip/hip_runtime.h"
/************************************************************************************************
* Implementing Singular Value Decomposition on GPU using CUDA using algorithm 			*
* given in IPDPS '09 paper "Singular Value Decomposition on GPU using CUDA"			*
*												*
* Copyright (c) 2009 International Institute of Information Technology, Hyderabad.		*
* All rights reserved.										*
*												*
* Permission to use, copy, modify and distribute this software and its documentation for 	*
* educational purpose is hereby granted without fee, provided that the above copyright		*
* notice and this permission notice appear in all copies of this software and that you do 	*
* not sell the software.									*
*												*
* THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND, EXPRESS, IMPLIED OR	*
* OTHERWISE.											*
* 												*
* Created by Sheetal Lahabar.									*
************************************************************************************************/
#ifndef _CUBIDIAGONAL_CU_
#define _CUBIDIAGONAL_CU_

#include "cubidiagonal.h"

int l = 0, m, n, count = 0;
int j = 0, k = 0, Nb, size = 0, i = 0, kMax = 0, var = 0;
int time1, time2, time3, time4;

struct timeb tr;

//CPU variables
float *d_Q, *d_P;
float *Wmat, *Zmat;
float *e1, *idenU, *idenV, *In;

//Device variables
float *d_Zmat, *d_Wmat, *d_Umat, *d_Vmat;
float *d_w1, *d_z1, *d_v1, *d_x1;

float *d_U, *d_V;
float *d_qw, *d_pw;

float *d_Aor, *d_W;
float *d_e1, *d_zinput;
float *d_iden, *iden1, *d_temp, *d_zero;

float dot, alpha = 0 , beta = 0;
float one = 1, s, t, d_eta, alpha1 = -1;
float norm = 0, eta, v = 0, g = -1, sigma = 0, value = 0;

hipblasStatus_t status1;

float sign(float eta)
{
	if(eta >= 0)
		return 1;
	else
		return -1;
}

float *iden(int m)
{
	int i = 0, j = 0, l = 0;
	float *a = (float*)malloc(sizeof(float) * m * m);

	for(i=0 ; i < m ; i++)
	{
		for(j=0 ; j < m ; j++)
		{
			if(i==j)
				a[l++]=1;
			else
				a[l++]=0;
		}
	}	
	return a;
}

float *zeros(int m)
{
	int i = 0;
	float *a = (float*)malloc(sizeof(float) * m);

	for(i=0 ; i < m ; i++)
		a[i] = 0;
	return a;
}

bool cubidiagonal(int M, int N, float *d_A, float *d_Q1, float *d_P1, double *diagonal, double *superdiag)
{
	bool result = 1;
 
	m = M;
	n = N;

//	if(n <= 128)
//		Nb = 1;
//	else
		Nb = 16;

	kMax = n/Nb;	

	idenU  = (float*)malloc(sizeof(float) * m * m);
	idenV  = (float*)malloc(sizeof(float) * n * n);

	Wmat   = (float*)malloc(sizeof(float) * m * m);
	Zmat   = (float*)malloc(sizeof(float) * n * Nb);

        In     = (float*)malloc(sizeof(float) * n * n);

	idenU  = iden(m);
	idenV  = iden(n);

	Wmat   = zeros(m * m);
	Zmat   = zeros(n * Nb);
	In     = zeros(n * n);
	
	if(m > n)
		size = m;
	else 
		size = n;

	iden1  = (float*)malloc(sizeof(float) * size);
	for(i=0; i < size ; i++)
		iden1[i] = 1;

	e1 = zeros(size);

	status1 = cublasAlloc(m*Nb, sizeof(d_Wmat[0]),   (void**)&d_Wmat);
	status1 = cublasAlloc(n*Nb, sizeof(d_Zmat[0]),   (void**)&d_Zmat);

	status1 = cublasAlloc(m*Nb,   sizeof(d_qw[0]),   (void**)&d_qw);
	status1 = cublasAlloc(n*Nb,   sizeof(d_pw[0]),   (void**)&d_pw);

	status1 = cublasAlloc( m*m,    sizeof(d_U[0]),    (void**)&d_U);
	status1 = cublasAlloc( n*n,    sizeof(d_V[0]),    (void**)&d_V);

	status1 = cublasAlloc( m*m, sizeof(d_Umat[0]),    (void**)&d_Umat);
	status1 = cublasAlloc( n*n, sizeof(d_Vmat[0]),    (void**)&d_Vmat);

	status1 = cublasAlloc( m*m,    sizeof(d_Q[0]),    (void**)&d_Q);
	status1 = cublasAlloc( n*n,    sizeof(d_P[0]),    (void**)&d_P);
	
	status1 = cublasAlloc(size,   sizeof(d_e1[0]),    (void**)&d_e1);
	status1 = cublasAlloc(size,   sizeof(d_w1[0]),    (void**)&d_w1);
	status1 = cublasAlloc(size,   sizeof(d_v1[0]),    (void**)&d_v1);
	status1 = cublasAlloc(size,   sizeof(d_x1[0]),    (void**)&d_x1);
	status1 = cublasAlloc(size,   sizeof(d_z1[0]),    (void**)&d_z1);
	status1 = cublasAlloc(size, sizeof(d_iden[0]),    (void**)&d_iden);
	status1 = cublasAlloc(size, sizeof(d_zero[0]),    (void**)&d_zero);
	status1 = cublasAlloc(size, sizeof(d_temp[0]),    (void**)&d_temp);
	status1 = cublasAlloc(size, sizeof(d_temp[0]),    (void**)&d_zinput);
	status1 = cublasAlloc(size, sizeof(d_temp[0]),    (void**)&d_Aor);
	status1 = cublasAlloc(size,    sizeof(d_W[0]),    (void**)&d_W);
	
	status1 = hipblasSetMatrix(m, m, sizeof(d_U[0]), Wmat, m, d_U, m);
	status1 = hipblasSetMatrix(n, n, sizeof(d_V[0]), In, n, d_V, n);

	status1 = hipblasSetMatrix(m, m, sizeof(d_Q[0]), idenU, m, d_Q, m);
	status1 = hipblasSetMatrix(n, n, sizeof(d_P[0]), idenV, n, d_P, n);

	status1 = hipblasSetMatrix(m, Nb, sizeof(d_qw[0]), Wmat , m, d_qw, m);
	status1 = hipblasSetMatrix(Nb, n, sizeof(d_pw[0]), Zmat, Nb, d_pw, Nb);

	status1 = hipblasSetMatrix(m, Nb, sizeof(d_Wmat[0]), Wmat, m, d_Wmat, m);
	status1 = hipblasSetMatrix(Nb, n, sizeof(d_Zmat[0]), Zmat, Nb, d_Zmat, Nb);

	status1 = hipblasSetMatrix(m, m, sizeof(d_Umat[0]), Wmat, m, d_Umat, m);
	status1 = hipblasSetMatrix(n, n, sizeof(d_Vmat[0]), In, n, d_Vmat, n);

	status1 = hipblasSetVector(size,   sizeof(d_e1[0]),    e1, 1,   d_e1, 1);
	status1 = hipblasSetVector(size, sizeof(d_iden[0]), iden1, 1, d_iden, 1);
	status1 = hipblasSetVector(size, sizeof(d_zero[0]),  Wmat, 1, d_zero, 1);
	status1 = hipblasSetVector(size, sizeof(d_zero[0]),  Wmat, 1,  d_Aor, 1);
	status1 = hipblasSetVector(size, sizeof(d_zero[0]),  Wmat, 1, d_temp, 1);
	
	printf("Intialization Complete\n");
	printf("Computing bidiagonal matrix B\n");

	ftime(&tr);
	time1 = tr.time;
	time2 = tr.millitm;

	for(i=0; i < kMax ; i++)
	{
		status1 = hipblasSetMatrix( m, Nb, sizeof(d_Wmat[0]), Wmat, m, d_Wmat, m );
		status1 = hipblasSetMatrix( Nb, n, sizeof(d_Zmat[0]), Zmat, Nb, d_Zmat, Nb );

		   norm = hipblasSnrm2( m - (Nb * i), &d_A[ m * Nb * i + Nb * i ] , 1 );
		status1 = hipblasGetVector( 1, sizeof(float), &d_A[ m * Nb * i + Nb * i ], 1, &eta,1 );

		v = sign(eta) * norm;
		var = -1 * sign(eta);

		if(v != 0)
			sigma = (eta + v)/v;
		else 
			sigma = 0;

		value = v / (eta + v);
		status1 = hipblasSetVector( 1, sizeof(float), &value, 1, &d_Umat[ m*Nb*i + Nb*i ], 1 );
		hipblasSaxpy( m - (Nb*i), 1/(eta+v), &d_A[ m * Nb *i + Nb * i ], 1, &d_Umat[ m * Nb * i + Nb * i ], 1 );

		//Q---
		hipblasSgemm( 'n', 'n', m, 1,  m - (Nb * i), sigma, &d_Q[ m * Nb * i ], m, &d_Umat[ m * Nb * i + Nb * i ], m, beta, &d_qw[0], m );
                hipblasSgemm( 'n', 'n', m, 1, 1, -1, &d_qw[0], m, &d_Umat[ m * Nb * i + Nb * i ], 1, 1, &d_Q[ m * Nb * i ] , m );
		//Q---

		//value = norm * g;
		value = norm * var;
		status1 = hipblasSetVector( 1 , sizeof(float), &value, 1, &d_A[ m * Nb * i + Nb * i ], 1 );

		hipblasSgemm( 't','n', n - (Nb*i+1), 1, m - (Nb*i), -1 * sigma, &d_A[ m * Nb * i + Nb * i + m ], m, &d_Umat[ m * Nb * i + Nb * i ], m, beta, &d_v1[0], size );
		hipblasSaxpy( n - (Nb*i+1), 1, &d_A[ m * Nb * i + Nb*i + m ], m, &d_v1[0], 1 );
		hipblasScopy( n - (Nb*i+1), &d_v1[0], 1, &d_temp[0], 1 );	
		
		norm = hipblasSnrm2( n - (Nb*i+1), &d_v1[0], 1 );
		status1 = hipblasGetVector( 1, sizeof(float), &d_v1[0], 1, &e1[0], 1 );
		eta = e1[0];

		hipblasSgemm( 't','t', n - (Nb*i+1), 1, 1 ,1, &d_A[ m * Nb * i + Nb*i + m ], m, &d_iden[0], n, -1, &d_v1[0], size );	
		hipblasScopy( n - (Nb*i+1), &d_v1[0], 1, &d_Zmat[ Nb * i * Nb + Nb ], Nb );

		v = sign(eta) * norm;
		var = -1 * sign(eta);	

		if(v !=0)
			sigma = (eta + v)/v;
		else
			sigma = 0;
	
		value = v / (eta + v);
		status1 = hipblasSetVector( 1 , sizeof(float), &value, 1, &d_Vmat[ n * Nb * i + Nb * i + n ], n );
		hipblasSaxpy( n - (Nb*i+1), 1/(eta+v), &d_temp[0], 1, &d_Vmat[ n * Nb * i + Nb * i + n ], n );

		status1 = hipblasSetVector( 1 , sizeof(float), &value, 1, &d_e1[0], 1 );
		hipblasSaxpy( n-(Nb*i+1), 1/(eta+v), &d_temp[0], 1, &d_e1[0], 1 );

		//P---
		hipblasSgemm( 't', 't', n, 1, n - (Nb*i+1), sigma, &d_P[ Nb * i + 1 ], n, &d_Vmat[ n * Nb * i + Nb * i + n ], n, beta, &d_W[0], size );
                hipblasScopy( n, &d_W[0], 1, &d_pw[0], Nb );
                hipblasSgemm( 'n', 't', 1, n, 1, -1, &d_Vmat[ n * Nb * i + Nb * i + n ], n, &d_W[0], size, 1, &d_P[ Nb * i +1 ], n ); 				//P---
		
		//value = norm * g;
		value = norm * var;

		hipblasSgemm( 'n', 't',  m - (Nb*i), 1, n - (Nb*i+1), sigma, &d_A[ m * Nb * i + Nb * i + m ], m , &d_Vmat[ n * Nb * i + Nb * i + n ], n, beta, &d_Wmat[ Nb * i ], m );

		if(m * Nb * i + Nb * i + m < (m * n))
			status1 = hipblasSetVector( 1, sizeof(float), &value, 1, &d_A[ m * Nb * i + Nb * i + m ], 1 );

		dot = hipblasSdot ( n - (Nb*i+1), &d_Zmat[ Nb * i * Nb + Nb ], Nb, &d_Vmat[ n * Nb * i + Nb * i + n ], n );
		dot = dot * sigma;
		hipblasSaxpy( n - (Nb*i+1), -1 * dot, &d_Vmat[ n * Nb * i + Nb * i + n ], n, &d_Zmat[ Nb * i * Nb + Nb ], Nb );

		alpha1 = -1;

		for(k=1 ; k < Nb ; k++)
		{
			hipblasScopy( size - (Nb * i) , &d_zero[0], 1, &d_e1[0], 1 );			

			hipblasSgemm( 'n', 'n', m - (Nb*i+k), 1, k, -1, &d_Umat[ Nb * m * i + Nb * i + k ], m, &d_Zmat[ Nb * i * Nb  + k * Nb ], Nb, 1, &d_A[ m * Nb * i + Nb * i + k * m + k ], m );
			hipblasSgemm( 'n', 'n', m - (Nb*i+k), 1, k, -1, &d_Wmat[ i * Nb + k ], m, &d_Vmat[ n * Nb * i + Nb * i  + k * n ], n, 1, &d_A[ m * Nb * i + Nb * i + k * m + k ], m );
		
			norm = hipblasSnrm2( m - (Nb*i+k) , &d_A[ m * Nb * i + Nb * i + k * m + k ] , 1 );
			status1 = hipblasGetVector( 1, sizeof(float), &d_A[ m * Nb * i + Nb * i + k * m + k ], 1, &eta, 1 );

			v = sign(eta) * norm;
			var = -1 * sign(eta);

			if(v!=0)
				sigma = (eta + v)/v;
			else
				sigma = 0;

			value = v / (eta + v);
			status1 = hipblasSetVector( 1 , sizeof(float), &value, 1, &d_e1[0], 1 );
			hipblasSaxpy( m - (Nb*i+k), 1/(eta+v), &d_A[ m * Nb * i + Nb * i + k * m + k ], 1, &d_e1[0], 1 );

			//value = norm * g;
			value = norm * var;

			status1 = hipblasSetVector( 1, sizeof(float), &value, 1, &d_A[ m * Nb * i + Nb * i + k * m + k ], 1 );
			hipblasScopy( m - (Nb*i+k), &d_e1[0], 1, &d_Umat[ m * Nb * i + Nb * i + k * m + k ], 1 );
			
			//Q---
			hipblasSgemm( 'n', 'n', m, 1, m - (Nb*i+k), sigma, &d_Q[ m * Nb * i + k * m ], m, &d_e1[0], size, beta, &d_W[0], size );
                        hipblasSgemm( 'n', 't', m, m - (Nb*i+k), k, 1, &d_qw[0], m, &d_Umat[ Nb * i * m + Nb * i + k ], m, beta, &d_U[0], m );
                        hipblasSgemm( 'n', 'n', m, 1, m - (Nb*i+k), -sigma, &d_U[0], m, &d_e1[0], size, 1, &d_W[0], size );

                        hipblasScopy( m, &d_W[0], 1, &d_qw[ k * m ], 1 );

                        hipblasSgemm( 'n', 't', m, 1, k, -1, &d_qw[0], m, &d_Umat[ m * Nb * i + Nb * i + k ], m, 1, &d_Q[ m * Nb * i + m * k ], m );
                        hipblasSgemm( 'n', 't', m, 1, 1, -1, &d_W[0], size, &d_e1[0], size, 1, &d_Q[ m * Nb * i + m * k ], m );	
			//Q---
			
			hipblasScopy( n - (Nb*i+k+1), &d_A[ m * Nb * i + Nb * i + k * m + k + m ], m, &d_temp[0], 1 );

			hipblasSgemm( 'n', 'n', 1, n - (Nb*i+k+1), k, -1, &d_Umat[ Nb * m * i + Nb * i + k ], m, &d_Zmat[ Nb * Nb  * i  + (k+1) * Nb ], Nb, 1, &d_A[ m * Nb * i + Nb * i + k * m + k + m ], m ); 
			hipblasSgemm( 'n', 'n', 1, n - (Nb*i+k+1), k ,-1, &d_Wmat[ Nb * i + k ], m, &d_Vmat[ n * Nb * i + Nb * i  + (k+1) * n ], n, 1, &d_A[ m * Nb * i + Nb * i + k * m + k + m ], m );

			float s1 = 1 - sigma; 

			hipblasScopy( n - (Nb*i+k+1), &d_A[ m * Nb * i + Nb * i + k * m + k + m ], m, &d_Aor[0], 1 );

			hipblasSgemm( 't', 'n', k, 1, m - (Nb*i+k+1), sigma, &d_Umat[ Nb * m * i + Nb * i + k + 1 ], m, &d_e1[1], size, 0, &d_U[0], m );
			hipblasSgemm( 't', 'n', n - (Nb*i+k+1), 1, k, 1, &d_Zmat[ Nb * Nb * i + (k+1) * Nb ], Nb, &d_U[0], m, 0, &d_V[0], n );	
			hipblasSgemm( 't', 'n', k, 1, m - (Nb*i+k+1), sigma, &d_Wmat[ Nb * i + k + 1 ], m, &d_e1[1], size, 0, &d_U[0], m );	
			hipblasSgemm( 't', 'n', n - (Nb*i+k+1), 1, k, 1, &d_Vmat[ n * Nb * i + Nb * i + (k+1)* n ], n, &d_U[0], m, 1, &d_V[0], n );

			hipblasSaxpy ( n - (Nb*i+k+1), s1, &d_A[ m * Nb * i + Nb * i + k * m + k + m ], m, &d_V[0], 1 );

			hipblasScopy( n - (Nb*i+k+1), &d_temp[0], 1, &d_A[ m * Nb * i + Nb * i + k * m + k + m ], m );
			hipblasSgemm( 't', 't', n - (Nb*i+k+1), 1, m - (Nb*i+k+1), -1 * sigma, &d_A[ m * Nb * i + Nb * i + (k + 1) * m + k + 1 ], m, &d_e1[1], 1, beta, &d_v1[0], size );

			hipblasSaxpy( n - (Nb*i+k+1), 1, &d_v1[0], 1, &d_V[0], 1 );
			hipblasScopy( n - (Nb*i+k+1), &d_V[0], 1, &d_temp[0], 1 );

			norm = hipblasSnrm2( n-(Nb*i+k+1), &d_temp[0], 1 );                           
			status1 = hipblasGetVector( 1, sizeof(float), &d_temp[0], 1, &e1[0], 1 );     
			eta = e1[0];                                                               
               											   
			v = sign(eta) * norm;
			var = -1 * sign(eta);

			if(v!=0)
				sigma = (eta + v)/v;
			else
				sigma = 0;

			value = v / (eta + v);

			hipblasScopy( size - (Nb * i), &d_zero[0], 1, &d_e1[0],  1 );			
			status1 = hipblasSetVector( 1, sizeof(float), &value, 1, &d_e1[0], 1 );
			hipblasSaxpy( n-(Nb*i+k+1), 1/(eta+v), &d_temp[0], 1, &d_e1[0], 1 );

			//P---
			if(Nb + 1 + Nb * (i-1) + k + 1 <= n)
			{
				hipblasSgemm( 't', 'n', n, 1, n - (Nb*i+k+1), sigma, &d_P[ Nb * i + k + 1], n, &d_e1[0], size, beta, &d_W[0], size );
                                hipblasSgemm( 't', 'n', n, n - (Nb*i+k+1), k, 1, &d_pw[0], Nb, &d_Vmat[ Nb * i * n + Nb * i + k * n + n ], n, beta, &d_V[0], n );
                                hipblasSgemm( 'n', 'n', n, 1, n - (Nb*i+k+1), -sigma, &d_V[0], n, &d_e1[0], size, 1, &d_W[0], size );

                                hipblasScopy( n, &d_W[0], 1, &d_pw[k], Nb );

                                hipblasSgemm( 't', 'n', 1, n, k, -1, &d_Vmat[ n * Nb * i + Nb * i + k * n + n ], n, &d_pw[0], Nb, 1, &d_P[ Nb * i + k + 1 ], n );
                                hipblasSgemm( 't', 't', 1, n, 1, -1, &d_e1[0], size, &d_W[0], size, 1, &d_P[ Nb * i + k + 1 ], n );
			}
			//P---

			
			hipblasSgemm( 'n', 't', m - (Nb*i+k+1), 1, n - (Nb*i+k+1), 1, &d_A[ m * Nb * i + Nb*i + (k+1) * m + k + 1 ], m, &d_e1[0] , 1, beta, &d_Wmat[ Nb * i + k * m + k + 1 ], m );
			hipblasSgemm( 'n', 't', k, 1, n - (Nb*i+k+1), 1, &d_Zmat[ Nb * Nb * i  + (k + 1)* Nb ], Nb, &d_e1[0], 1, 0, &d_U[0], m );
			hipblasSgemm( 'n', 'n', m - (Nb*i+k+1), 1, k, -sigma, &d_Umat[ m * Nb * i + Nb * i + k + 1 ], m, &d_U[0], m, sigma, &d_Wmat[ Nb * i + k * m + k + 1 ], m );

			dot  =  sigma * hipblasSdot ( n-(Nb*i+k+1), &d_Aor[0], 1, &d_e1[0], 1 );

			hipblasSgemm( 'n', 't', k, 1, n - (Nb*i+k+1), 1, &d_Vmat[ n * Nb * i  + Nb * i + (k + 1) * n ], n, &d_e1[0], 1, 0, &d_U[0], m );
			hipblasSgemm( 'n', 'n', m - (Nb*i+k+1), 1, k, -sigma, &d_Wmat[ Nb * i + k + 1 ], m, &d_U[0], m, 1, &d_Wmat[ Nb * i + k * m + k + 1 ], m );


			status1 = hipblasSetVector( 1, sizeof(float), &dot, 1, &d_Wmat[ Nb * i + k * m + k ], 1 );
			hipblasSgemm( 'n', 'n', n - (Nb*i+k+1), 1, 1, 1, &d_Aor[0], size, &d_iden[0], 1, -1, &d_temp[0], size ) ;


			hipblasScopy( n - (Nb*i+k+1), &d_temp[0], 1, &d_Zmat[ Nb * i * Nb +  Nb * (k+1)  + k ], Nb );		 			     		hipblasScopy( n - (Nb*i+k+1), &d_e1[0], 1, &d_Vmat[ n * Nb * i + Nb * i + (k + 1) * n + k ], n );
                        
			//value = norm * g;
			value = norm * var;
						
			if( m * Nb * i + Nb * i + (k + 1) * m + k < (m * n) ) 
				status1 = hipblasSetVector( 1 , sizeof(float), &value, 1, &d_A[ m * Nb * i + Nb * i + (k + 1)* m + k ], 1 );

			dot = hipblasSdot ( n - (Nb*i+k+1), &d_Zmat[ Nb * i * Nb +  Nb * (k+1)  + k ], Nb, &d_e1[0], 1 );
			dot = dot * -sigma;
			e1[0] = dot;

			hipblasSaxpy( n - (Nb*i+k+1), e1[0], &d_e1[0], 1, &d_Zmat[ Nb * i * Nb +  Nb * (k+1)  + k ], Nb );
		}

		//Update A
		hipblasSgemm( 'n', 'n', m - (Nb * (i+1)), n - (Nb * (i+1)), Nb, -1, &d_Umat[ Nb * i * m + Nb * i + Nb ], m, &d_Zmat[ Nb * Nb * (i+1) ], Nb, 1, &d_A[ m * Nb * (i+1)  + Nb * (i+1) ], m ) ;	
		hipblasSgemm( 'n', 'n', m - (Nb * (i+1)), n - (Nb * (i+1)), Nb, -1, &d_Wmat[ Nb * (i + 1) ], m, &d_Vmat[ Nb * n * i + n * Nb  + Nb * i ], n, 1, &d_A[ m * Nb * (i+1)  + Nb * (i+1) ], m );

		//Update Q
		hipblasSgemm( 'n', 't', m, m - (Nb * (i+1)), Nb, -1, &d_qw[0], m, &d_Umat[ m * Nb * i + Nb * i + Nb ], m, 1, &d_Q[ m * Nb * (i+1) ], m );
		//Update P
                if( n - (Nb+1) - Nb * (i-1) -1  >= Nb )                 
                        hipblasSgemm( 't', 'n', n - (Nb * (i+1) + 1), n, Nb, -1, &d_Vmat[ n * Nb * (i+1) + Nb * i + n ], n, &d_pw[0], Nb, 1, &d_P[ Nb * (i+1) + 1], n );
      	}
	
	hipDeviceSynchronize();
	ftime(&tr);
	time3 = tr.time;
	time4 = tr.millitm;

	printf("Bidiagonalization complete: Time required %d %d \n", time3 - time1, time4 - time2);

	float *tempA;
	tempA = (float*)malloc(sizeof(float)*m*n);

	status1 = hipblasGetMatrix(m, n, sizeof(float), &d_A[0], m, tempA, m);

	//Copy diagonal elements to CPU
        for(j=0; j < n ; j++)
                diagonal[j] = (double)tempA[(m+1)*j];

	//Copy superdiagonal elements to CPU
	for(j=0; j < n-1; j++)
		superdiag[j] = (double)tempA[m + (m+1)*j];

	CUDA_SAFE_CALL(hipFree(d_w1));
	CUDA_SAFE_CALL(hipFree(d_z1));
	CUDA_SAFE_CALL(hipFree(d_v1));
	CUDA_SAFE_CALL(hipFree(d_x1));
	CUDA_SAFE_CALL(hipFree(d_e1));

	CUDA_SAFE_CALL(hipFree(d_Umat));
	CUDA_SAFE_CALL(hipFree(d_Vmat));
	CUDA_SAFE_CALL(hipFree(d_Zmat));
	CUDA_SAFE_CALL(hipFree(d_Wmat));

	CUDA_SAFE_CALL(hipFree(d_U));
	CUDA_SAFE_CALL(hipFree(d_V));

	CUDA_SAFE_CALL(hipFree(d_qw));
	CUDA_SAFE_CALL(hipFree(d_pw));

	CUDA_SAFE_CALL(hipFree(d_Aor));
	CUDA_SAFE_CALL(hipFree(d_W));
	CUDA_SAFE_CALL(hipFree(d_iden));
	CUDA_SAFE_CALL(hipFree(d_zero));
	CUDA_SAFE_CALL(hipFree(d_temp));
	CUDA_SAFE_CALL(hipFree(d_zinput));

	return result;
}
#endif 
